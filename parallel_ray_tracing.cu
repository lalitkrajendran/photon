#include "hip/hip_runtime.h"
/*
 * parallel_ray_tracing.cu
 *
 *  Created on: Apr 20, 2016
 *      Author: lrajendr
 */
#include <stdio.h>
#include <fstream>
#include <string>
#include "parallel_ray_tracing.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include "float3_operators.h"
#include <iostream>
#include <stdlib.h>
#include <numeric>
using namespace std;
//#incldue <cutil_math.h>

hipArray *data_array = 0;
texture<float, 2> mie_scattering_irradiance;

__device__ float random_single(unsigned int seed)
{

  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t state;

  /* the seed can be the same for each core, here we pass the time in from the CPU */
  /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
  /* the offset is how much extra we advance in the sequence for each call, can be 0 */

  /* we have to initialize the state */
  hiprand_init(seed, blockIdx.x, 0, &state);

  float rand_num = hiprand_uniform(&state);
  return rand_num;
}

__global__ void generate_lightfield_angular_data(float lens_pitch, float image_distance,
		scattering_data_t scattering_data, int scattering_type, lightfield_source_t lightfield_source,
                                     int lightray_number_per_particle, int n_min, int n_max, float beam_wavelength,
                                     float aperture_f_number, light_ray_data_t light_ray_data)
{
	/*
		This function generates the light field data for the source points specified by the
		structure lightfield_source.  The data is only generated for the source points from
		n_min to n_max.  The parameter lightray_number_per_particle is the number of rays to generate for each
		source point.
	*/

	//--------------------------------------------------------------------------------------
	// compute indices to access in lightfield_source and lightfield_data
	//--------------------------------------------------------------------------------------

	// find global thread ID
//	int block_id = blockIdx.x;
	int local_thread_id = threadIdx.x;
//	int global_thread_id = block_id + local_thread_id;

	float del_scattering_angle = (scattering_data.scattering_angle[1] - scattering_data.scattering_angle[0])*180.0/M_PI;
	float min_scattering_angle = scattering_data.scattering_angle[0];

	// get id of particle which is the source of light rays
	int particle_id = blockIdx.x*blockDim.x + local_thread_id;

	// get id of ray emitted by the particle
	int local_ray_id = blockIdx.z;
	int global_ray_id = local_ray_id + particle_id*lightray_number_per_particle;

	if(global_ray_id >= lightfield_source.num_particles*lightray_number_per_particle)
		return;

	// get source coordinates of the light ray
	float x_current = lightfield_source.x[particle_id];;
	float y_current = lightfield_source.y[particle_id];;
	float z_current = lightfield_source.z[particle_id];;

	//--------------------------------------------------------------------------------------
	// compute direction of propagation of light ray
	//--------------------------------------------------------------------------------------

	// generate random points on the lens
	float random_number_1 = random_single(particle_id * local_ray_id * global_ray_id);
	float random_number_2 = random_single(particle_id + local_ray_id + global_ray_id);
	float x_lens = 0.5*lens_pitch*random_number_1*cos(2*M_PI*random_number_2);
	float y_lens = 0.5*lens_pitch*random_number_1*sin(2*M_PI*random_number_2);

	// calculate the x angles for the light rays
	float theta_temp = -(x_lens - x_current) / (image_distance - z_current);
	// calculate the y angles for the light rays
	float phi_temp = -(y_lens - y_current) / (image_distance - z_current);



	//--------------------------------------------------------------------------------------
	// compute irradiance of the light ray
	//--------------------------------------------------------------------------------------

	int diameter_index;
	float3 ray_direction_vector, temp_vector;
	float dot_vector[3];
	float ray_scattering_angles, ray_scattering_irradiance;
	double irradiance_current;
	float3 beam_propogation_vector;
	// if scattering_type is mie, then use mie scattering data
	if(scattering_type)
	{
		//% This extracts the normalized beam propagation direction vector from the
		//% parameters structure
		beam_propogation_vector.x=scattering_data.beam_propogation_vector[0];
		beam_propogation_vector.y=scattering_data.beam_propogation_vector[1];
		beam_propogation_vector.z=scattering_data.beam_propogation_vector[2];

		// % This extracts the current particle diameter index
		diameter_index = lightfield_source.diameter_index[particle_id];
		// % This calculates the light ray's direction vectors (in the camera
		// % coordinate system)
		ray_direction_vector = make_float3(x_lens-x_current,y_lens-y_current,image_distance-z_current);

		// % This normalizes the ray direction vectors
		ray_direction_vector = normalize(ray_direction_vector);

		// % This rotates the light rays direction vectors by the inverse of the
        // % camera rotation array so that the ray is now in the world coordinate
        // % system
		for(int i = 0; i < 3; i++)
		{
			temp_vector.x = scattering_data.inverse_rotation_matrix[i*3 + 0];
			temp_vector.y = scattering_data.inverse_rotation_matrix[i*3 + 1];
			temp_vector.z = scattering_data.inverse_rotation_matrix[i*3 + 2];

			dot_vector[i] = dot(temp_vector,ray_direction_vector);
		}
		ray_direction_vector = make_float3(dot_vector[0],dot_vector[1],dot_vector[2]);

		// % This calculates the angle that the light ray direction vectors make
		// % with the laser propagation direction vector in radians
		ray_scattering_angles = angleBetween(beam_propogation_vector,ray_direction_vector);
		// % This calculates the Mie scattering irradiance at the current
		// % scattered angle and with the current particle diameter
		int lookup_angle = (int)(ray_scattering_angles - scattering_data.scattering_angle[0])/del_scattering_angle;
		ray_scattering_irradiance = scattering_data.scattering_irradiance[lookup_angle*27 + diameter_index];

//		ray_scattering_irradiance = tex2D(mie_scattering_irradiance,diameter_index,lookup_angle);
		// % This calculates the total irradiance for the current particle's rays
		irradiance_current=ray_scattering_irradiance*lightfield_source.radiance[particle_id];
	}
	// if not mie scattering, then set irradiance to be uniform
	else
	{
		// % This specifies the total irradiance for the current particle's
		// % rays to be uniform
		irradiance_current = lightfield_source.radiance[particle_id];
	}

	// save the light rays to the light field data structure
	light_ray_data.ray_source_coordinates[global_ray_id] = make_float3(x_current,y_current,z_current);
	light_ray_data.ray_propagation_direction[global_ray_id] = normalize(make_float3(theta_temp,phi_temp,1.0));
	light_ray_data.ray_wavelength[global_ray_id] = beam_wavelength;
	light_ray_data.ray_radiance[global_ray_id] = 1/(aperture_f_number*aperture_f_number)*irradiance_current;
//	d_lightfield_data.x[global_ray_id] = x_current;
//	d_lightfield_data.y[global_ray_id] = y_current;
//	d_lightfield_data.z[global_ray_id] = z_current;
//	d_lightfield_data.theta[global_ray_id] = theta_temp;
//	d_lightfield_data.phi[global_ray_id] = phi_temp;
//	d_lightfield_data.radiance[global_ray_id] = irradiance_current;

}


int main(int argc, char** argv)
{

	printf("Hello world\n");
//	float lens_pitch,image_distance;
//	scattering_data_t* scattering_data_p;
//	char* scattering_type_str;
//	lightfield_source_t* lightfield_source_p;
//	int lightray_number_per_particle, n_min, n_max;
//	lightfield_data_t* lightfield_data_p;
//
//	start_ray_tracing(lens_pitch,image_distance,scattering_data_p,scattering_type_str,
//			lightfield_source_p, lightray_number_per_particle,n_min,n_max,lightfield_data_p);

	return 0;
}

//int add(int a, int b)
//{
//	return a+b;
//}


extern "C"{

void read_from_file()
{
	float lens_pitch, image_distance, beam_wavelength, aperture_f_number;
	scattering_data_t scattering_data_p;
	int scattering_type;
	lightfield_source_t lightfield_source_p;
	int lightray_number_per_particle;
	int n_min; int n_max;

	/*
	 * 	This function saves the data passed from python to a file.
	 * 	This is done to enable debugging this program within eclipse.
	 */

	int k,l;

	//--------------------------------------------------------------------------------------
	// save scalars to file
	//--------------------------------------------------------------------------------------

	// open file
	string filename = "/home/barracuda/a/lrajendr/Projects/parallel_ray_tracing/data/scalars.bin";
	std::ifstream file_scalars(filename.c_str(), std::ios::in |
				std::ios::binary);
	// lens_pitch
	file_scalars.read((char*)&lens_pitch, sizeof(float));

	// image_distance
	file_scalars.read((char*)&image_distance, sizeof(float));

	// scattering_type
	file_scalars.read((char*)&scattering_type, sizeof(int));

	// n_min
	file_scalars.read((char*)&n_min, sizeof(int));

	// n_max
	file_scalars.read((char*)&n_max, sizeof(int));

	// lightray_number_per_particle
	file_scalars.read((char*)&lightray_number_per_particle, sizeof(int));

	// beam_wavelength
	file_scalars.read((char*)&beam_wavelength, sizeof(float));

	// aperture_f_number
	file_scalars.read((char*)&aperture_f_number, sizeof(float));

	file_scalars.close();

	//--------------------------------------------------------------------------------------
	// read scattering data
	//--------------------------------------------------------------------------------------

	// open file
	filename = "/home/barracuda/a/lrajendr/Projects/parallel_ray_tracing/data/scattering_data.bin";
	std::ifstream file_scattering(filename.c_str(), std::ios::in |
			std::ios::binary);
	// inverse rotation matrix
	for(k = 0; k < 9; k++)
		file_scattering.read ((char*)&scattering_data_p.inverse_rotation_matrix[k], sizeof(float));


	// beam_propogation_vector
	for(k = 0; k < 3; k++)
		file_scattering.read ((char*)&scattering_data_p.beam_propogation_vector[k], sizeof(float));

	// num_angles
	file_scattering.read ((char*)&scattering_data_p.num_angles, sizeof(int));

	// num_diameters
	file_scattering.read ((char*)&scattering_data_p.num_diameters, sizeof(int));

	// scattering_angle
	scattering_data_p.scattering_angle = (float *) malloc(scattering_data_p.num_angles*sizeof(float));
	for(k = 0; k < scattering_data_p.num_angles; k++)
			file_scattering.read ((char*)&scattering_data_p.scattering_angle[k], sizeof(float));

	// scattering_irradiance
	scattering_data_p.scattering_irradiance = (float *) malloc(scattering_data_p.num_angles * scattering_data_p.num_diameters*sizeof(float));
	for(k = 0; k < scattering_data_p.num_angles * scattering_data_p.num_diameters; k++)
			file_scattering.read ((char*)&scattering_data_p.scattering_irradiance[k], sizeof(float));

	file_scattering.close();

	//--------------------------------------------------------------------------------------
	// save lightfield_source data to file
	//--------------------------------------------------------------------------------------

	// open file
	filename = "/home/barracuda/a/lrajendr/Projects/parallel_ray_tracing/data/lightfield_source.bin";
	std::ifstream file_lightfield_source(filename.c_str(), std::ios::in |
			std::ios::binary);

	// lightray_number_per_particle
	file_lightfield_source.read ((char*)&lightfield_source_p.lightray_number_per_particle, sizeof(int));

	// lightray_process_number
	file_lightfield_source.read ((char*)&lightfield_source_p.lightray_process_number, sizeof(int));

	// num_particles
	file_lightfield_source.read ((char*)&lightfield_source_p.num_particles, sizeof(int));

	// num_rays
	file_lightfield_source.read ((char*)&lightfield_source_p.num_rays, sizeof(int));

	// diameter_index
	lightfield_source_p.diameter_index = (int *) malloc(lightfield_source_p.num_particles * sizeof(int));
	for(k = 0; k < lightfield_source_p.num_particles; k++)
		file_lightfield_source.read ((char*)&lightfield_source_p.diameter_index[k], sizeof(int));

	// radiance
	lightfield_source_p.radiance = (double *) malloc(lightfield_source_p.num_particles * sizeof(double));
	for(k = 0; k < lightfield_source_p.num_particles; k++)
		file_lightfield_source.read ((char*)&lightfield_source_p.radiance[k], sizeof(double));

	// x
	lightfield_source_p.x = (float *) malloc(lightfield_source_p.num_particles*sizeof(float));
	for(k = 0; k < lightfield_source_p.num_particles; k++)
		file_lightfield_source.read ((char*)&lightfield_source_p.x[k], sizeof(float));

	// y
	lightfield_source_p.y = (float *) malloc(lightfield_source_p.num_particles*sizeof(float));
	for(k = 0; k < lightfield_source_p.num_particles; k++)
		file_lightfield_source.read ((char*)&lightfield_source_p.y[k], sizeof(float));

	// z
	lightfield_source_p.z = (float *) malloc(lightfield_source_p.num_particles*sizeof(float));
	for(k = 0; k < lightfield_source_p.num_particles; k++)
		file_lightfield_source.read ((char*)&lightfield_source_p.z[k], sizeof(float));

	file_lightfield_source.close();


//	char* scattering_type_str;
//	if(scattering_type)
//		strcpy(scattering_type_str,"mie");
//	else
//		strcpy(scattering_type_str,"diffuse");
	char scattering_type_str[] = "mie";

	start_ray_tracing(lens_pitch, image_distance,&scattering_data_p, scattering_type_str,&lightfield_source_p,lightray_number_per_particle,n_min, n_max,beam_wavelength,aperture_f_number);


}

void save_to_file(float lens_pitch, float image_distance,
		scattering_data_t* scattering_data_p, char* scattering_type_str,
		lightfield_source_t* lightfield_source_p, int lightray_number_per_particle,
		int n_min, int n_max,float beam_wavelength, float aperture_f_number)
{
	/*
	 * 	This function saves the data passed from python to a file.
	 * 	This is done to enable debugging this program within eclipse.
	 */

	int k,l;
	int scattering_type = strcmp(scattering_type_str,"mie")==0 ? 1 : 0;
	//--------------------------------------------------------------------------------------
	// save scalars to file
	//--------------------------------------------------------------------------------------
	printf("saving scalars to file\n");
	// open file
	string filename = "/home/barracuda/a/lrajendr/Projects/parallel_ray_tracing/data/scalars.bin";
	std::ofstream file_scalars(filename.c_str(), std::ios::out |
				std::ios::binary);
	// lens_pitch
	file_scalars.write((char*)&lens_pitch, sizeof(float));

	// image_distance
	file_scalars.write((char*)&image_distance, sizeof(float));

	// scattering_type
	file_scalars.write((char*)&scattering_type, sizeof(int));

	// n_min
	file_scalars.write((char*)&n_min, sizeof(int));

	// n_max
	file_scalars.write((char*)&n_max, sizeof(int));

	// lightray_number_per_particle
	file_scalars.write((char*)&lightray_number_per_particle, sizeof(int));

	// beam_wavelength
	file_scalars.write((char*)&beam_wavelength, sizeof(float));

	// aperture_f_number
	file_scalars.write((char*)&aperture_f_number, sizeof(float));

	file_scalars.close();

	//--------------------------------------------------------------------------------------
	// save scattering data to file
	//--------------------------------------------------------------------------------------
	if(strcmp(scattering_type_str,"mie")==0){

	// open file
	filename = "/home/barracuda/a/lrajendr/Projects/parallel_ray_tracing/data/scattering_data.bin";
	std::ofstream file_scattering(filename.c_str(), std::ios::out |
			std::ios::binary);
	// inverse rotation matrix
	printf("inverse rotation matrix: \n");
	for(k = 0; k < 9; k++){
		if(k%3==0)
			printf("\n");
		printf("%f ",scattering_data_p->inverse_rotation_matrix[k]);
		scattering_data_p->inverse_rotation_matrix[k] = (float) scattering_data_p->inverse_rotation_matrix[k];
			file_scattering.write ((char*)&scattering_data_p->inverse_rotation_matrix[k], sizeof(float));

//		}
		printf("\n");
	}
	printf("\n");
	// beam_propogation_vector
	printf("beam propagation vector\n");
	for(k = 0; k < 3; k++){
		printf("%f ", scattering_data_p->beam_propogation_vector[k]);
		file_scattering.write ((char*)&scattering_data_p->beam_propogation_vector[k], sizeof(float));
	}
	// num_angles
	file_scattering.write ((char*)&scattering_data_p->num_angles, sizeof(int));
	// num_diameters
	file_scattering.write ((char*)&scattering_data_p->num_diameters, sizeof(int));
	// scattering_angle
	for(k = 0; k < scattering_data_p->num_angles; k++)
			file_scattering.write ((char*)&scattering_data_p->scattering_angle[k], sizeof(float));
	// scattering_irradiance
	for(k = 0; k < scattering_data_p->num_angles * scattering_data_p->num_diameters; k++)
			file_scattering.write ((char*)&scattering_data_p->scattering_irradiance[k], sizeof(float));

	file_scattering.close();
	}
	//--------------------------------------------------------------------------------------
	// save lightfield_source data to file
	//--------------------------------------------------------------------------------------

	// open file
	filename = "/home/barracuda/a/lrajendr/Projects/parallel_ray_tracing/data/lightfield_source.bin";
	std::ofstream file_lightfield_source(filename.c_str(), std::ios::out |
			std::ios::binary);

	// lightray_number_per_particle
	file_lightfield_source.write ((char*)&lightfield_source_p->lightray_number_per_particle, sizeof(int));
	// lightray_process_number
	file_lightfield_source.write ((char*)&lightfield_source_p->lightray_process_number, sizeof(int));
	// num_particles
	file_lightfield_source.write ((char*)&lightfield_source_p->num_particles, sizeof(int));
	// num_rays
	file_lightfield_source.write ((char*)&lightfield_source_p->num_rays, sizeof(int));
	// diameter_index
	for(k = 0; k < lightfield_source_p->num_particles; k++)
		file_lightfield_source.write ((char*)&lightfield_source_p->diameter_index[k], sizeof(int));
	// radiance
	for(k = 0; k < lightfield_source_p->num_particles; k++)
		file_lightfield_source.write ((char*)&lightfield_source_p->radiance[k], sizeof(double));
	// x
	for(k = 0; k < lightfield_source_p->num_particles; k++)
		file_lightfield_source.write ((char*)&lightfield_source_p->x[k], sizeof(float));
	// y
		for(k = 0; k < lightfield_source_p->num_particles; k++)
			file_lightfield_source.write ((char*)&lightfield_source_p->y[k], sizeof(float));
	// z
	for(k = 0; k < lightfield_source_p->num_particles; k++)
		file_lightfield_source.write ((char*)&lightfield_source_p->z[k], sizeof(float));

	file_lightfield_source.close();


}

int add(int a, int b)
{
	return a+b;
}

void start_ray_tracing(float lens_pitch, float image_distance,
		scattering_data_t* scattering_data_p, char* scattering_type_str,
		lightfield_source_t* lightfield_source_p, int lightray_number_per_particle,
		int n_min, int n_max,float beam_wavelength, float aperture_f_number)
{
	// create instance of structure using the pointers
	scattering_data_t scattering_data = *scattering_data_p;
	lightfield_source_t lightfield_source = *lightfield_source_p;

	int source_point_number = n_max - n_min + 1;

	// allocate space for the light field variables on the CPU

	int N = lightray_number_per_particle*source_point_number;

	//--------------------------------------------------------------------------------------
	// allocate space on GPU for lightfield_source
	//--------------------------------------------------------------------------------------

	// declare pointers to device arrays
	float* d_source_x;
	float* d_source_y;
	float* d_source_z;
	double *d_source_radiance;
	int *d_source_diameter_index;
	int num_particles = lightfield_source.num_particles;

	// allocate space for device arrays on GPU
	//hipMalloc((void **)&gpuData, sizeof(float)*size);
	float *gpuData;
	int size = 10;
	hipDeviceSynchronize();
	hipMalloc((void **)&gpuData, sizeof(float)*size);
	hipMalloc((void **)&d_source_x,sizeof(float)*num_particles);
	hipMalloc((void **)&d_source_y,num_particles*sizeof(float));
	hipMalloc((void **)&d_source_z,num_particles*sizeof(float));
	hipMalloc((void **)&d_source_radiance,num_particles*sizeof(double));
	hipMalloc((void **)&d_source_diameter_index,num_particles*sizeof(int));

	// copy data to GPU
	hipMemcpy(d_source_x,lightfield_source.x,num_particles*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_source_y,lightfield_source.y,num_particles*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_source_z,lightfield_source.z,num_particles*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_source_radiance,lightfield_source.radiance,num_particles*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_source_diameter_index,lightfield_source.diameter_index,num_particles*sizeof(int),hipMemcpyHostToDevice);

	// make copy of host structure
	lightfield_source_t  lightfield_source_copy = lightfield_source;

	// point host structure to device array
	lightfield_source.x = d_source_x;
	lightfield_source.y = d_source_y;
	lightfield_source.z = d_source_z;
	lightfield_source.radiance = d_source_radiance;
	lightfield_source.diameter_index = d_source_diameter_index;

	//--------------------------------------------------------------------------------------
	// allocate space on GPU for scattering_data
	//--------------------------------------------------------------------------------------

	// declare pointers to device arrays
	float *d_scattering_angle;
	float* d_scattering_irradiance;

	// allocate space for device arrays on GPU
	hipMalloc((void**)&d_scattering_angle,scattering_data.num_angles*sizeof(float));
	hipMalloc((void**)&d_scattering_irradiance,scattering_data.num_angles*scattering_data.num_diameters*sizeof(float));

	// copy data to GPU
	hipMemcpy(d_scattering_angle,scattering_data.scattering_angle,scattering_data.num_angles*sizeof(float)
	,hipMemcpyHostToDevice);
	hipMemcpy(d_scattering_irradiance,scattering_data.scattering_irradiance,scattering_data.num_angles*scattering_data.num_diameters*sizeof(float)
		,hipMemcpyHostToDevice);

	// make copy of host structure
	scattering_data_t scattering_data_copy = scattering_data;

	// point host structure to device array
	scattering_data.scattering_angle = d_scattering_angle;

	hipMalloc((void**)&data_array,scattering_data.num_angles*scattering_data.num_diameters*sizeof(float));
	hipMemcpy(data_array,scattering_data.scattering_irradiance,
				scattering_data.num_angles*scattering_data.num_diameters*sizeof(float),hipMemcpyHostToDevice);
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipBindTexture2D( NULL, mie_scattering_irradiance,
	                               data_array,
	                               desc, scattering_data.num_diameters, scattering_data.num_angles,
	                               sizeof(float) * scattering_data.num_diameters );
	scattering_data.scattering_irradiance = d_scattering_irradiance;

	//--------------------------------------------------------------------------------------
	// allocate space on GPU for light_ray_data
	//--------------------------------------------------------------------------------------

	// allocate space for light_ray_data on CPU
	light_ray_data_t light_ray_data;
	light_ray_data.ray_source_coordinates = (float3 *) malloc(N*sizeof(float3));
	light_ray_data.ray_propagation_direction = (float3 *) malloc(N*sizeof(float3));
	light_ray_data.ray_wavelength = (float *) malloc(N*sizeof(float));
	light_ray_data.ray_radiance = (double *) malloc(N*sizeof(double));
	light_ray_data.num_lightrays = N;
	// declare pointers to GPU arrays
	float3 *d_ray_source_coordinates, *d_ray_propagation_direction;
	float *d_ray_wavelength;
	double *d_ray_radiance;

	// allocate memory on GPU
	hipMalloc((void**)&d_ray_source_coordinates, N*sizeof(float3));
	hipMalloc((void**)&d_ray_propagation_direction, N*sizeof(float3));
	hipMalloc((void**)&d_ray_wavelength, N*sizeof(float));
	hipMalloc((void**)&d_ray_radiance, N*sizeof(double));

	// initialize arrays to zero
	hipMemset(d_ray_source_coordinates,0.0,N*sizeof(float3));
	hipMemset(d_ray_propagation_direction,0.0,N*sizeof(float3));
	hipMemset(d_ray_wavelength,0.0,N*sizeof(float));
	hipMemset(d_ray_radiance,0.0,N*sizeof(double));

	// copy contents of light_ray_data structure
	light_ray_data_t light_ray_data_copy = light_ray_data;
	// point structure to device arrays
	light_ray_data.ray_source_coordinates = d_ray_source_coordinates;
	light_ray_data.ray_propagation_direction = d_ray_propagation_direction;
	light_ray_data.ray_wavelength = d_ray_wavelength;
	light_ray_data.ray_radiance = d_ray_radiance;

	int scattering_type = 0;
	if(strcmp(scattering_type_str,"mie")==0)
		scattering_type = 1;


	// allocate threads per block
	dim3 block(10,1,1);
	// allocate blocks per grid
	dim3 grid(source_point_number/block.x,1,lightray_number_per_particle);

	// call kernel
	generate_lightfield_angular_data<<<grid,block>>>(lens_pitch, image_distance,scattering_data,
			scattering_type, lightfield_source,lightray_number_per_particle, n_min, n_max,
			beam_wavelength,aperture_f_number,light_ray_data);

	hipDeviceSynchronize();

	// copy light_ray_data back to host
	hipMemcpy(light_ray_data_copy.ray_source_coordinates,light_ray_data.ray_source_coordinates,N*sizeof(float3),hipMemcpyDeviceToHost);
	hipMemcpy(light_ray_data_copy.ray_propagation_direction,light_ray_data.ray_propagation_direction,N*sizeof(float3),hipMemcpyDeviceToHost);
	hipMemcpy(light_ray_data_copy.ray_wavelength,light_ray_data.ray_wavelength,N*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(light_ray_data_copy.ray_radiance,light_ray_data.ray_radiance,N*sizeof(double),hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	// point original light_ray_data to updated host arrays
	light_ray_data.ray_source_coordinates = light_ray_data_copy.ray_source_coordinates;
	light_ray_data.ray_propagation_direction = light_ray_data_copy.ray_propagation_direction;
	light_ray_data.ray_wavelength = light_ray_data_copy.ray_wavelength;
	light_ray_data.ray_radiance = light_ray_data_copy.ray_radiance;

	// display first and last few elements of lightfield_data
	N = light_ray_data.num_lightrays;
	printf("lightfield_data contents\n");
	printf("ray_source_coordinates (1st): %f, %f, %f\n",light_ray_data.ray_source_coordinates[0].x,light_ray_data.ray_source_coordinates[0].y,light_ray_data.ray_source_coordinates[0].z);
	printf("ray_source_coordinates (last): %f, %f, %f\n",light_ray_data.ray_source_coordinates[N-1].x,light_ray_data.ray_source_coordinates[N-1].y,light_ray_data.ray_source_coordinates[N-1].z);
	printf("ray_propagation_direction (1st): %f, %f, %f\n",light_ray_data.ray_propagation_direction[0].x,light_ray_data.ray_propagation_direction[0].y,light_ray_data.ray_propagation_direction[0].z);
	printf("ray_propagation_direction (last): %f, %f, %f\n",light_ray_data.ray_propagation_direction[N-1].x,light_ray_data.ray_propagation_direction[N-1].y,light_ray_data.ray_propagation_direction[N-1].z);
	printf("ray_wavelength (1st, last): %f, %f\n",light_ray_data.ray_wavelength[0],light_ray_data.ray_wavelength[N-1]);
	printf("ray_radiance (1st, last): %f, %f\n",light_ray_data.ray_radiance[0],light_ray_data.ray_radiance[N-1]);


	// free pointers
	hipFree(gpuData);
	hipFree(d_source_x);
	hipFree(d_source_y);
	hipFree(d_source_z);
	hipFree(d_source_radiance);
	hipFree(d_source_diameter_index);

	hipFree(d_scattering_angle);
	hipFree(d_scattering_irradiance);
	hipFree(data_array);

	hipFree(d_ray_source_coordinates);
	hipFree(d_ray_propagation_direction);
	hipFree(d_ray_wavelength);
	hipFree(d_ray_radiance);

}



}


