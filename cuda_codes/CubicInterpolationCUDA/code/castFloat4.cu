#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.

When using this code in a scientific project, please cite one or all of the
following papers:
*  Daniel Ruijters and Philippe Th�venaz,
   GPU Prefilter for Accurate Cubic B-Spline Interpolation, 
   The Computer Journal, vol. 55, no. 1, pp. 15-20, January 2012.
   http://dannyruijters.nl/docs/cudaPrefilter3.pdf
*  Daniel Ruijters, Bart M. ter Haar Romeny, and Paul Suetens,
   Efficient GPU-Based Texture Interpolation using Uniform B-Splines,
   Journal of Graphics Tools, vol. 13, no. 4, pp. 61-69, 2008.
\*--------------------------------------------------------------------------*/

#ifndef _CAST_FLOAT4_H_
#define _CAST_FLOAT4_H_

#include "memcpy.cu"


//--------------------------------------------------------------------------
// Declare the interleaved copu CUDA kernel
//--------------------------------------------------------------------------
template<class T> __global__ void CopyCastInterleaved(uchar* destination, const T* source, uint pitch, uint width)
{
	uint2 index = make_uint2(
		__umul24(blockIdx.x, blockDim.x) + threadIdx.x,
		__umul24(blockIdx.y, blockDim.y) + threadIdx.y);
	uint index3 = 3 * (index.y * width + index.x);
	
	float4* dest = (float4*)(destination + index.y * pitch) + index.x;
	float mult = 1.0f / Multiplier<T>();
	*dest = make_float4(
		mult * (float)source[index3],
		mult * (float)source[index3+1],
		mult * (float)source[index3+2], 1.0f);
}

//--------------------------------------------------------------------------
// Declare the typecast templated function
// This function can be called directly in C++ programs
//--------------------------------------------------------------------------

//! Allocate GPU memory and copy a voxel volume from CPU to GPU memory
//! and cast it to the normalized floating point format
//! @return the pointer to the GPU copy of the voxel volume
//! @param host  pointer to the voxel volume in CPU (host) memory
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
template<class T> extern hipPitchedPtr CastVolumeHost3ToDevice4(const T* host, uint width, uint height, uint depth)
{
	hipPitchedPtr device = {0};
	const hipExtent extent = make_hipExtent(width * sizeof(float4), height, depth);
	CUDA_SAFE_CALL(hipMalloc3D(&device, extent));
	const size_t pitchedBytesPerSlice = device.pitch * device.ysize;
	
	T* temp = 0;
	const uint voxelsPerSlice = width * height;
	const size_t nrOfBytesTemp = voxelsPerSlice * 3 * sizeof(T);
	CUDA_SAFE_CALL(hipMalloc((void**)&temp, nrOfBytesTemp));

	uint dimX = min(PowTwoDivider(width), 64);
	dim3 dimBlock(dimX, min(PowTwoDivider(height), 512 / dimX));
	dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
	size_t offsetHost = 0;
	size_t offsetDevice = 0;
	
	for (uint slice = 0; slice < depth; slice++)
	{
		CUDA_SAFE_CALL(hipMemcpy(temp, host + offsetHost, nrOfBytesTemp, hipMemcpyHostToDevice));
		CopyCastInterleaved<T><<<dimGrid, dimBlock>>>((uchar*)device.ptr + offsetDevice, temp, (uint)device.pitch, width);
		CUT_CHECK_ERROR("Cast kernel failed");
		offsetHost += voxelsPerSlice;
		offsetDevice += pitchedBytesPerSlice;
	}

	CUDA_SAFE_CALL(hipFree(temp));  //free the temp GPU volume
	return device;
}

#endif  //_CAST_FLOAT4_H_
