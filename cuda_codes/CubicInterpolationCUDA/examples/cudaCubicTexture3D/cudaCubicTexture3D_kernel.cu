#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2009, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.

When using this code in a scientific project, please cite one or all of the
following papers:
*  Daniel Ruijters and Philippe Th�venaz,
   GPU Prefilter for Accurate Cubic B-Spline Interpolation, 
   The Computer Journal, vol. 55, no. 1, pp. 15-20, January 2012.
   http://dannyruijters.nl/docs/cudaPrefilter3.pdf
*  Daniel Ruijters, Bart M. ter Haar Romeny, and Paul Suetens,
   Efficient GPU-Based Texture Interpolation using Uniform B-Splines,
   Journal of Graphics Tools, vol. 13, no. 4, pp. 61-69, 2008.
\*--------------------------------------------------------------------------*/

#include <stdio.h>
#include <cutil.h>
#include <memcpy.cu>
#include <cubicPrefilter3D.cu>
#include <cubicTex3D.cu>

texture<uchar, 3, hipReadModeNormalizedFloat> tex;  //3D texture
texture<float, 3, hipReadModeElementType> coeffs;  //3D texture


__global__ void
render_kernel(uchar* output, uint2 imageExtent, float3 volumeExtent, float w, uint filterMethod)
{
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	float u = x / (float)imageExtent.x;
	float v = y / (float)imageExtent.y;
	float3 coord = volumeExtent * make_float3(u, v, w);

	// read from 3D texture
	float voxel;
	switch (filterMethod)
	{
		case 0:  //nearest neighbor
		case 1: voxel = linearTex3D(tex, coord); break;  //linear
		case 2: voxel = cubicTex3DSimple(coeffs, coord); break;  //simple cubic
		case 3: voxel = cubicTex3D(coeffs, coord); break;  //fast cubic
		case 4: voxel = cubicTex3D(tex, coord); break;  //non-prefiltered, fast cubic
	}

	// write output color
	uint i = __umul24(y, imageExtent.x) + x;
	output[i] = __saturatef(voxel) * 255;
}


// render image using CUDA
extern "C" void render(uchar* output, uint2 imageExtent, uint3 volumeSize, float w, uint filterMethod)
{
	// set texture parameters
	tex.filterMode = (filterMethod == 0) ? hipFilterModePoint : hipFilterModeLinear;

	// call CUDA kernel, writing results to PBO
	const dim3 blockSize(min(PowTwoDivider(imageExtent.x), 16), min(PowTwoDivider(imageExtent.y), 16));
	const dim3 gridSize(imageExtent.x / blockSize.x, imageExtent.y / blockSize.y);
	const float3 volumeExtent = make_float3((float)volumeSize.x, (float)volumeSize.y, (float)volumeSize.z);
	render_kernel<<<gridSize, blockSize>>>(output, imageExtent, volumeExtent, w, filterMethod);
	CUT_CHECK_ERROR("kernel failed");
}


// intialize the textures, and calculate the cubic B-spline coefficients
extern "C" void initCuda(const uchar* voxels, uint3 volumeSize)
{
	// calculate the b-spline coefficients
	hipPitchedPtr bsplineCoeffs = CastVolumeHostToDevice(voxels, volumeSize.x, volumeSize.y, volumeSize.z);
	CubicBSplinePrefilter3DTimer((float*)bsplineCoeffs.ptr, (uint)bsplineCoeffs.pitch, volumeSize.x, volumeSize.y, volumeSize.z);

	// create the b-spline coefficients texture
	hipArray *coeffArray = 0;
	hipExtent volumeExtent = make_hipExtent(volumeSize.x, volumeSize.y, volumeSize.z);
	CreateTextureFromVolume(&coeffs, &coeffArray, bsplineCoeffs, volumeExtent, true);
	CUDA_SAFE_CALL(hipFree(bsplineCoeffs.ptr));  //they are now in the coeffs texture, we do not need this anymore

	// Now create a texture with the original sample values for nearest neighbor and linear interpolation
	// Note that if you are going to do cubic interpolation only, you can remove the following code
	hipArray *volumeArray = 0;
	CreateTextureFromVolume(&tex, &volumeArray, voxels, volumeExtent, false);
}
